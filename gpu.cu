#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include "info.h"
#include "gpu.h"

typedef struct board {
  int cells[BOARD_DIM * BOARD_DIM];
} board_t;

__device__ int status_d = 0;

__global__ void check_cell(board_t* board) {
  int x = threadIdx.x;
  int y = threadIdx.y;

  // Calculate current cell id in the board
  int cell_id = y * BOARD_DIM + x;

  // Get the current cell
  int cur_cell = board->cells[cell_id];

  // Flag the current cell as checked 
  // 1 - unchecked
  // 0 - checked
  int cur_check = 1;

  // Keep checking the cell until all cells have been checked
  while (__syncthreads_or(cur_check) != 0) {  
    cur_check = 0;
    if (cur_cell == 0) {
      continue;
    }

    // Check row success
    int row_s = cur_cell;
    for (int c = x - 2; c <= x + 2; c++) {
      if (c < 0 || c >= BOARD_DIM ||
          board->cells[y * BOARD_DIM + c] != cur_cell) {
        row_s = 0;
      }
    }

    // Check col success
    int col_s = cur_cell;
    for (int r = y - 2; r <= y + 2; r++) {
      if (r < 0 || r >= BOARD_DIM ||
          board->cells[r * BOARD_DIM + x] != cur_cell) {
        col_s = 0;
      }
    }

    // Check / success
    int rl_s = cur_cell;
    for (int i = -2; i <= 2; i++) {
      int c = x + i;
      int r = y - i;
      if (c < 0 || c >= BOARD_DIM ||
          r < 0 || r >= BOARD_DIM ||
          board->cells[r * BOARD_DIM + c] != cur_cell) {
        rl_s = 0;
      }
    }

    // Check \ success
    int lr_s = cur_cell;
    for (int i = - 2; i <= 2; i++) {
      int c = x + i;
      int r = y + i;
      if (c < 0 || c >= BOARD_DIM ||
          r < 0 || r >= BOARD_DIM ||
          board->cells[r * BOARD_DIM + c] != cur_cell) {
        lr_s = 0;
      }
    }

    // Check complete
    int complete = 4;
    for (int r = 0; r < BOARD_DIM; r++) {
      for(int c = 0; c < BOARD_DIM; c++) {
        if (board->cells[r * BOARD_DIM + c] == 0) {
          complete = 0;
        }
      }
    }

    // Compile results
    int cur_ret = row_s | col_s | rl_s | lr_s | complete;
    if (complete == 4) {
      status_d = complete;
      return;
    } else if (cur_ret != 0) {
      status_d = cur_cell;
      return;
    }
  }
  return;
}


void check_board(int** raw_board, int* res) {
  // Parse raw board
  board_t* board = (board_t*) malloc(sizeof(board_t));
  for(int i = 0; i < BOARD_DIM; i++) {
    for(int j = 0; j < BOARD_DIM; j++) {
      board->cells[i*BOARD_DIM + j] = raw_board[i][j];
    }
  }

  // Malloc memory in gpu
  board_t* gpu_board;
  if (hipMalloc(&gpu_board, sizeof(board_t)) != hipSuccess) {
    fprintf(stderr, "Failed to allocate the board\n");
    exit(2);
  }

  // Copy board to gpus
  if(hipMemcpy(gpu_board, board, sizeof(board_t), hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy board to the GPU\n");
    exit(2);
  }

  // Solve the boards
  check_cell<<<1,dim3(BOARD_DIM, BOARD_DIM)>>>(gpu_board);

  // Wait until it is finished
  if(hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(hipPeekAtLastError()));
    exit(2);
  }

  // Update status
  int status_h;
  hipMemcpyFromSymbol(&status_h, HIP_SYMBOL(status_d), sizeof(status_d), 0, hipMemcpyDeviceToHost);

  // Free the gpu memory
  hipFree(gpu_board);
  *res = status_h;
  //fprintf(stderr, "Current result is %d\n", *res);
}

/*
//Test
int main(int argc, char** argv) {
  int** testboard;
  testboard = (int **)malloc(BOARD_DIM * sizeof(int*)); 
  for (int i=0; i<BOARD_DIM; i++) 
    testboard[i] = (int*)calloc(BOARD_DIM, sizeof(int)); 
  testboard[2][4] = 1;
  testboard[3][4] = 1;
  testboard[4][4] = 1;
  testboard[5][4] = 1;
  testboard[6][4] = 1;

  int res;
  check_board(testboard, &res);
  printf("this is a test in main function. Winner is %d\n", res);
  return 0;
}*/
